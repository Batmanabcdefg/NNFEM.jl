#include "hip/hip_runtime.h"
__global__ void forward_(double *out, const double *y, const double *H0, int n){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i>=n) return;
    double V[3];
    double v = 0.0;
    for(int i_=0;i_<3;i_++){
      V[i_] = H0[3*i_]*y[3*i] + H0[3*i_+1]*y[3*i+1] + H0[3*i_+2]*y[3*i+2];
    }

    for (int i_=0;i_<3;i_++){
          v += y[3*i+i_] * V[i_];
      }

    for(int i_=0;i_<3;i_++){
      for(int j_=0;j_<3;j_++){
        out[9*i+i_*3+j_] = H0[3*i_+j_] - V[i_]*V[j_]/(1.0+v);
      }
    }
}

__global__ void backward_(double *d_y, const double *d_out, const double *y, const double *H0, int n){
  double dO_dy, P[3][3], V[3];
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i>=n) return;
  double v = 0.0;
  for(int i_=0;i_<3;i_++){
    V[i_] = (H0[3*i_]*y[3*i] + H0[3*i_+1]*y[3*i+1] + H0[3*i_+2]*y[3*i+2]);
  }

  for (int i_=0;i_<3;i_++){
        v += y[3*i+i_] * V[i_];
    }

  for(int i_=0;i_<3;i_++){
    V[i_] /= (1 + v);
  }

  for(int i_=0;i_<3;i_++){
    for(int j_=0;j_<3;j_++){
      P[i_][j_] = V[i_]*V[j_];
    }
  }
  for (int k=0; k<3; k++){ 
    d_y[3*i+k] = 0.0;
    double val = 2*(H0[3*k]*y[3*i] + H0[3*k+1]*y[3*i+1] + H0[3*k+2]*y[3*i+2]);
    for(int i_=0;i_<3;i_++){
      for(int j_=0;j_<3;j_++){
        dO_dy = - (H0[3*k+i_]*V[j_] + V[i_]*H0[3*k+j_]) + P[i_][j_]*val;
        d_y[3*i+k] += dO_dy*d_out[9*i+3*i_+j_];
      }
    }
    
  }  
}




void forwardGPU(double *out, const double *y, const double *H0, int n){
  forward_<<<(n+255)/256, 256>>>(out, y, H0, n);
}


void backwardGPU(double *d_y, const double *d_out, const double *y, const double *H0, int n){
  backwardGPU<<<(n+255)/256, 256>>>(d_y, d_out, y, H0, n);
}
